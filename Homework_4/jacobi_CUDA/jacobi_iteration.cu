#include "hip/hip_runtime.h"
/* Host code for the Jacobi method of solving a system of linear equations
 * by iteration.

 * Build as follws: make clean && make

 * Author: Naga Kandasamy
 * Date modified: May 21, 2020
*/

#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "jacobi_iteration.h"

/* Include the kernel code */
#include "jacobi_iteration_kernel.cu"

/* Uncomment the line below if you want the code to spit out debug information. */
/* #define DEBUG */

int main(int argc, char **argv)
{
	if (argc > 1) {
		printf("This program accepts no arguments\n");
		exit(EXIT_FAILURE);
	}

	matrix_t  A;                    /* N x N constant matrix */
	matrix_t  B;                    /* N x 1 b matrix */
	matrix_t reference_x;           /* Reference solution */
	matrix_t gpu_naive_solution_x;  /* Solution computed by naive kernel */
  matrix_t gpu_opt_solution_x;    /* Solution computed by optimized kernel */

	/* Initialize the random number generator */
	srand(time(NULL));

	/* Generate diagonally dominant matrix */
  printf("\nGenerating %d x %d system\n", MATRIX_SIZE, MATRIX_SIZE);
	A = create_diagonally_dominant_matrix(MATRIX_SIZE, MATRIX_SIZE);
	if (A.elements == NULL) {
		printf("Error creating matrix\n");
    exit(EXIT_FAILURE);
	}

  /* Create the other vectors */
  B = allocate_matrix_on_host(MATRIX_SIZE, 1, 1);
	reference_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
	gpu_naive_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
  gpu_opt_solution_x = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);

	#ifdef DEBUG
		print_matrix(A);
		print_matrix(B);
		print_matrix(reference_x);
	#endif

	struct timeval start, stop;

	gettimeofday(&start, NULL);
  /* Compute Jacobi solution on CPU */
	printf("\nPerforming Jacobi iteration on the CPU\n");
  compute_gold(A, reference_x, B);
  display_jacobi_solution(A, reference_x, B); /* Display statistics */
	gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time for CPU = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
										(stop.tv_usec - start.tv_usec) / (float)1000000));

	/* Compute Jacobi solution on device. Solutions are returned in gpu_naive_solution_x and gpu_opt_solution_x. */
  printf("\nPerforming Jacobi iteration on device\n");
	compute_on_device(A, gpu_naive_solution_x, gpu_opt_solution_x, B);
	printf("\nShowing results for gpu_naive_solution\n");
  display_jacobi_solution(A, gpu_naive_solution_x, B); /* Display statistics */
	printf("\nShowing results for gpu_opt_solution\n");
  display_jacobi_solution(A, gpu_opt_solution_x, B);

  free(A.elements);
	free(B.elements);
	free(reference_x.elements);
	free(gpu_naive_solution_x.elements);
  free(gpu_opt_solution_x.elements);

  exit(EXIT_SUCCESS);
}


/* FIXME: Complete this function to perform Jacobi calculation on device */
void compute_on_device(const matrix_t A, matrix_t gpu_naive_sol_x, matrix_t gpu_opt_sol_x, const matrix_t B)
{
	int done = 0;
	int num_iter = 0;
	double ssd, mse;

	double *d_ssd = NULL; /* Pointer to device address holding ssd */

	/* Allocate matrices to hold iteration values */
	matrix_t new_x_naive = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);
	matrix_t new_x_opt = allocate_matrix_on_host(MATRIX_SIZE, 1, 0);

	struct timeval start, stop;

	/* initialize solution of x for GPU */
	for (unsigned int i = 0; i < A.num_rows; i++){
		float e = B.elements[i];
		gpu_naive_sol_x.elements[i] = e;
		gpu_opt_sol_x.elements[i] = e;
	}

	/* Allocating space on device for matricies on the GPU with error checking */
	matrix_t device_A = allocate_matrix_on_device(A);
	matrix_t device_naive_sol_x = allocate_matrix_on_device(gpu_naive_sol_x);
	matrix_t device_opt_sol_x = allocate_matrix_on_device(gpu_opt_sol_x);
	matrix_t device_B = allocate_matrix_on_device(B);
	matrix_t device_new_x_naive = allocate_matrix_on_device(new_x_naive);
	matrix_t device_new_x_opt = allocate_matrix_on_device(new_x_opt);

	/* Copying matricies A, B, and x solutions to GPU with error checking */
	copy_matrix_to_device(device_A, A);
	copy_matrix_to_device(device_B, B);
	copy_matrix_to_device(device_naive_sol_x, gpu_naive_sol_x);;
	copy_matrix_to_device(device_opt_sol_x, gpu_opt_sol_x);

	/* Allocating space for the device ssd on the GPU */
	hipMalloc((void**) &d_ssd, sizeof(double));

	/* Allocating space for the lock and initializing  mutex/locks on the GPU */
	int *mutex_on_device = NULL;
	hipMalloc((void **) &mutex_on_device, sizeof(int));
	hipMemset(mutex_on_device, 0, sizeof(int));

	printf("\nPerforming Jacobi Naive \n");
	/* Setting up the execution configuration for the naive kernel */
	dim3 threadevice_Block(1, THREAdevice_BLOCK_SIZE, 1);
	dim3 grid(1, (A.num_rows + THREAdevice_BLOCK_SIZE - 1)/ THREAdevice_BLOCK_SIZE);

	gettimeofday(&start, NULL);
	while (!done){
		hipMemset(d_ssd, 0.0, sizeof(double));

		/* using jacboi iteration kernel naive */
		jacobi_iteration_kernel_naive<<<grid, threadevice_Block>>>(device_A, device_naive_sol_x, device_new_x_naive, device_B, mutex_on_device, d_ssd);
		hipDeviceSynchronize();
		check_CUDA_error("KERNEL FAILURE: jacobi_iteration_kernel_naive\n");

		jacobi_update_x<<<grid,threadevice_Block>>>(device_naive_sol_x, device_new_x_naive);
		hipDeviceSynchronize();
		check_CUDA_error("KERNEL FAILURE: jacobi_update_x");

		/* Check for convergence and update the unknowns. */
		hipMemcpy(&ssd, d_ssd, sizeof(double), hipMemcpyDeviceToHost);
		num_iter++;
		mse = sqrt(ssd); /* Mean squared error. */

		if (mse <= THRESHOLD){
			done = 1;
			printf ("\nConvergence achieved after %d iterations \n", num_iter);
		}
		// printf ("Iteration: %d. MSE = %f\n", num_iter, mse);
	}
	gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time for GPU-Naive = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
										(stop.tv_usec - start.tv_usec) / (float)1000000));


	printf("\nPerforming Jacobi Optimized \n");
	/* Jacobi optimized kernel */
	threadevice_Block.x = threadevice_Block.y = TILE_SIZE;
	grid.x = 1;
	grid.y = (gpu_opt_sol_x.num_rows + TILE_SIZE - 1)/TILE_SIZE;

	done = 0;
	num_iter = 0;

	gettimeofday(&start, NULL);
	while (!done){
		hipMemset(d_ssd, 0.0, sizeof(double));

		/* using jacboi iteration kernel optimized */
		jacobi_iteration_kernel_optimized<<<grid, threadevice_Block>>>(device_A, device_opt_sol_x, device_new_x_opt, device_B, mutex_on_device, d_ssd);
        hipDeviceSynchronize();
				check_CUDA_error("KERNEL FAILURE: jacobi_iteration_kernel_optimized\n");

		jacobi_update_x<<<grid,threadevice_Block>>>(device_opt_sol_x, device_new_x_opt);
        hipDeviceSynchronize();
				check_CUDA_error("KERNEL FAILURE: jacobi_update_x");

        /* Check for convergence and update the unknowns. */
        hipMemcpy(&ssd, d_ssd, sizeof (double), hipMemcpyDeviceToHost);
        num_iter++;
        mse = sqrt(ssd);

      	if (mse <= THRESHOLD){
            done = 1;
            printf ("\nConvergence achieved after %d iterations \n", num_iter);
		}
		// printf ("Iteration: %d. MSE = %f\n", num_iter, mse);
	}
	gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time for GPU-Optimized = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
										(stop.tv_usec - start.tv_usec)/(float)1000000));

	/* Copying the solutions back from GPU */
	copy_matrix_from_device(gpu_naive_sol_x, device_naive_sol_x);
	check_CUDA_error("Copying matrix device_naive_sol_x from device");
	copy_matrix_from_device(gpu_opt_sol_x, device_opt_sol_x);
	check_CUDA_error("Copying matrix device_opt_sol_x from device");

	/* Freeing memory on GPU/ Clean up device memory */
	hipFree(device_A.elements);
	hipFree(device_B.elements);
	hipFree(device_naive_sol_x.elements);
	hipFree(device_opt_sol_x.elements);
	hipFree(d_ssd);

	hipFree(mutex_on_device);
	hipFree(device_new_x_naive.elements);
	hipFree(device_new_x_opt.elements);

	free(new_x_naive.elements);
	free(new_x_opt.elements);

  return;
}

/* Allocate matrix on the device of same size as M */
matrix_t allocate_matrix_on_device(const matrix_t M)
{
    matrix_t Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void **)&Mdevice.elements, size);
    return Mdevice;
}

/* Allocate a matrix of dimensions height * width.
   If init == 0, initialize to all zeroes.
   If init == 1, perform random initialization.
*/
matrix_t allocate_matrix_on_host(int num_rows, int num_columns, int init)
{
    matrix_t M;
    M.num_columns = num_columns;
    M.num_rows = num_rows;
    int size = M.num_rows * M.num_columns;

	M.elements = (float *)malloc(size * sizeof(float));
	for (unsigned int i = 0; i < size; i++) {
		if (init == 0)
            M.elements[i] = 0;
		else
            M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}

    return M;
}

/* Copy matrix to device */
void copy_matrix_to_device(matrix_t Mdevice, const matrix_t Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
    return;
}

/* Copy matrix from device to host */
void copy_matrix_from_device(matrix_t Mhost, const matrix_t Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
    return;
}

/* Prints the matrix out to screen */
void print_matrix(const matrix_t M)
{
	for (unsigned int i = 0; i < M.num_rows; i++) {
        for (unsigned int j = 0; j < M.num_columns; j++) {
			printf("%f ", M.elements[i * M.num_columns + j]);
        }

        printf("\n");
	}

    printf("\n");
    return;
}

/* Returns a floating-point value between [min, max] */
float get_random_number(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
	return (float)floor((double)(min + (max - min + 1) * r));
}

/* Check for errors in kernel execution */
void check_CUDA_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err) {
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    return;
}

/* Create diagonally dominant matrix */
matrix_t create_diagonally_dominant_matrix(unsigned int num_rows, unsigned int num_columns)
{
	matrix_t M;
	M.num_columns = num_columns;
	M.num_rows = num_rows;
	unsigned int size = M.num_rows * M.num_columns;
	M.elements = (float *)malloc(size * sizeof(float));
    if (M.elements == NULL)
        return M;

	/* Create a matrix with random numbers between [-.5 and .5] */
    unsigned int i, j;
	for (i = 0; i < size; i++)
        M.elements[i] = get_random_number (MIN_NUMBER, MAX_NUMBER);

	/* Make diagonal entries large with respect to the entries on each row. */
	for (i = 0; i < num_rows; i++) {
		float row_sum = 0.0;
		for (j = 0; j < num_columns; j++) {
			row_sum += fabs(M.elements[i * M.num_rows + j]);
		}

        M.elements[i * M.num_rows + i] = 0.5 + row_sum;
	}

    return M;
}
