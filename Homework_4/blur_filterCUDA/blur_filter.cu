#include "hip/hip_runtime.h"
/* Reference code implementing the box blur filter.

    Build and execute as follows:
        make clean && make
        ./blur_filter size

    Author: Naga Kandasamy
    Date created: May 3, 2019
    Date modified: May 12, 2020

    FIXME: Minjae Park & John Truong
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

/* #define DEBUG */

/* Include the kernel code */
#include "blur_filter_kernel.cu"

/* setting number of threwads and block size */
#define NUM_THREAD_BLOCKS 240
#define THREAD_BLOCK_SIZE 128

extern "C" void compute_gold(const image_t, image_t);
void compute_on_device(const image_t, image_t);
int check_results(const float *, const float *, int, float);
void print_image(const image_t);
image_t allocate_image_on_device (const image_t);
void copy_image_to_device(image_t, image_t);
void copy_image_from_device(image_t, image_t);
void check_CUDA_error(const char *);

int main(int argc, char **argv)
{
    if (argc < 2) {
        fprintf(stderr, "Usage: %s size\n", argv[0]);
        fprintf(stderr, "size: Height of the image. The program assumes size x size image.\n");
        exit(EXIT_FAILURE);
    }

    /* Allocate memory for the input and output images */
    int size = atoi(argv[1]);

    fprintf(stderr, "Creating %d x %d images\n", size, size);
    image_t in, out_gold, out_gpu;
    in.size = out_gold.size = out_gpu.size = size;
    in.element = (float *)malloc(sizeof(float) * size * size);
    out_gold.element = (float *)malloc(sizeof(float) * size * size);
    out_gpu.element = (float *)malloc(sizeof(float) * size * size);
    if ((in.element == NULL) || (out_gold.element == NULL) || (out_gpu.element == NULL)) {
        perror("Malloc");
        exit(EXIT_FAILURE);
    }

    /* Poplulate our image with random values between [-0.5 +0.5] */
    srand(time(NULL));
    int i;
    for (i = 0; i < size * size; i++)
        in.element[i] = rand()/(float)RAND_MAX -  0.5;

    struct timeval start, stop;

   /* Calculate the blur on the CPU. The result is stored in out_gold. */
    fprintf(stderr, "Calculating blur on the CPU\n");
    gettimeofday(&start, NULL);
    compute_gold(in, out_gold);
    gettimeofday(&stop, NULL);
    fprintf ("Execution time for CPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec +\
                  (stop.tv_usec - start.tv_usec)/(float)1000000));

#ifdef DEBUG
   print_image(in);
   print_image(out_gold);
#endif

   /* FIXME: Calculate the blur on the GPU. The result is stored in out_gpu. */
   fprintf(stderr, "Calculating blur on the GPU\n");
   compute_on_device(in, out_gpu);

   /* Check CPU and GPU results for correctness */
   fprintf(stderr, "Checking CPU and GPU results\n");
   int num_elements = out_gold.size * out_gold.size;
   float eps = 1e-6;    /* Do not change */
   int check;
   check = check_results(out_gold.element, out_gpu.element, num_elements, eps);
   if (check == 0)
       fprintf(stderr, "TEST PASSED\n");
   else
       fprintf(stderr, "TEST FAILED\n");

   /* Free data structures on the host */
   free((void *)in.element);
   free((void *)out_gold.element);
   free((void *)out_gpu.element);

    exit(EXIT_SUCCESS);
}

/* FIXME: Complete this function to calculate the blur on the GPU */
void compute_on_device(const image_t in, image_t out)
{
	/* Allocate memory on device for image */
	image_t d_in = allocate_image_on_device(in);
	image_t d_out = allocate_image_on_device(out);

	/* Copy image to memory of device */
	copy_image_to_device(d_in, in);

	/* Set up execution grid on the GPU */
  int num_thread_blocks = NUM_THREAD_BLOCKS;
	dim3 thread_block(THREAD_BLOCK_SIZE, 1, 1); /* Set number of threads in the thread block */
  fprintf(stderr, "Setting up a (%d x %d) execution grid\n", num_thread_blocks, num_thread_blocks);
  dim3 grid(NUM_THREAD_BLOCKS,1);

  fprintf(stderr, "\nKernel uses only gobal memory\n");
  struct timeval start, stop;
	gettimeofday(&start, NULL);
  /* Launch kernel with multiple thread blocks. The kernel call is non-blocking. */
	blur_filter_kernel<<<grid, thread_block>>>(d_in.element, d_out.element, d_in.size);
	gettimeofday(&stop, NULL);
	fprintf ("Execution time for GPU = %fs. \n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

  /* check for errors */
  check_CUDA_error("Error in kernel");

	/* Copy image out back over */
	copy_image_from_device(out, d_out);

	/* Free memory on GPU */
	hipFree(d_in.element);
	hipFree(d_out.element);

    return;
}

/* Allocate image on device */
image_t allocate_image_on_device (const image_t img)
{
	image_t img_device = img;
	int size = img.size * img.size * sizeof(float);

	hipMalloc((void**)&img_device.element, size);
  if(img_device.element == NULL){
    fprintf(stderr, "CudaMalloc error\n");
    exit(EXIT_FAILURE);
  }

	return img_device;
}

/* Copy image from host memory to device memory */
void copy_image_to_device(image_t img_device, const image_t img_host)
{
	int size = (img_host.size * img_host.size) * sizeof (float);
	hipMemcpy(img_device.element, img_host.element, size, hipMemcpyHostToDevice);
}

/* Copy image from device memory to host memory */
void copy_image_from_device(image_t img_host, image_t img_device)
{
	int size = (img_device.size * img_device.size) * sizeof (float);
	hipMemcpy(img_host.element, img_device.element, size, hipMemcpyDeviceToHost);
}

/* Check for errors during kernel execution */
void check_CUDA_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

/* Check correctness of results */
int check_results(const float *pix1, const float *pix2, int num_elements, float eps)
{
    int i;
    for (i = 0; i < num_elements; i++)
        if (fabsf((pix1[i] - pix2[i])/pix1[i]) > eps)
            return -1;

    return 0;
}

/* Print out the image contents */
void print_image(const image_t img)
{
    int i, j;
    float val;
    for (i = 0; i < img.size; i++) {
        for (j = 0; j < img.size; j++) {
            val = img.element[i * img.size + j];
            printf("%0.4f ", val);
        }
        printf("\n");
    }

    printf("\n");
}
