#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void
blur_filter_kernel (const float *in, float *out, int size)
{
  int pix;
  int i, j;
  int row, col;
  int curr_row, curr_col;
  float blur_value;
  int num_neighbors;

  /* Obtain thread index within the thread block */
  int threadX = threadIdx.x;

  /* Obtain block index within the grid */
  int blockX = blockIdx.x;

  /* Find position in matrix*/
  int idx = blockDim.x * blockX + threadX;

  /* Find striding position in matrix */
  int stride = blockDim.x * gridDim.x;

  for (pix = idx; pix < size * size; pix += stride)
  { /* Iterate over pixels in image */
      row = pix/size;             /* Obtain row number of pixel */
      col = pix % size;           /* Obtain column number of pixel */

      /* Apply blur filter to current pixel */
      blur_value = 0.0;
      num_neighbors = 0;
      for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++)
      {
          for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++)
          {
              /* Accumulate values of neighbors while checking for
               * boundary conditions */
              curr_row = row + i;
              curr_col = col + j;
              if ((curr_row > -1) && (curr_row < size) && (curr_col > -1) && (curr_col < size)) {
                  blur_value += in[curr_row * size + curr_col];
                  num_neighbors += 1;
              }
          }
      }

      /* Write averaged blurred value out */
      out[pix] = blur_value/num_neighbors;
  }
    return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
