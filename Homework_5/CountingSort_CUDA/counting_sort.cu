#include "hip/hip_runtime.h"
/* Host-side code to perform counting sort
 * Author: Naga Kandasamy
 * Date modified: May 27, 2020
 *
 * Compile as follows: make clean && make
 */

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <math.h>
#include <limits.h>
#include "counting_sort.h"
#include "counting_sort_kernel.cu"

/* Uncomment to spit out debug info */
// #define DEBUG

extern "C" int counting_sort_gold(int *, int *, int, int);
int rand_int(int, int);
void print_array(int *, int);
void print_min_and_max_in_array(int *, int);
void compute_on_device(int *, int *, int, int);
int check_if_sorted(int *, int);
int compare_results(int *, int *, int);
void check_for_error(const char *);

int main(int argc, char **argv)
{
    if (argc < 2) {
        printf("Usage: %s num-elements\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    struct timeval start, stop;
    int num_elements = atoi(argv[1]);
    int range = MAX_VALUE - MIN_VALUE;
    int *input_array, *sorted_array_reference, *sorted_array_d;

    /* Populate input array with random integers between [0, RANGE] */
    printf("Generating input array with %d elements in the range 0 to %d\n", num_elements, range);
    input_array = (int *)malloc(num_elements * sizeof(int));
    if (input_array == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }

    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++)
        input_array[i] = rand_int (MIN_VALUE, MAX_VALUE);

#ifdef DEBUG
    print_array(input_array, num_elements);
    print_min_and_max_in_array(input_array, num_elements);
#endif

    /* Sort elements in input array using reference implementation.
     * The result is placed in sorted_array_reference. */
    printf("\nSorting array on CPU\n");
    int status;
    sorted_array_reference = (int *)malloc(num_elements * sizeof(int));
    if (sorted_array_reference == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    gettimeofday(&start, NULL);
    memset(sorted_array_reference, 0, num_elements);
    status = counting_sort_gold(input_array, sorted_array_reference, num_elements, range);
    if (status == -1) {
        exit(EXIT_FAILURE);
    }
    gettimeofday(&stop, NULL);
    fprintf(stderr, "Execution time for CPU = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
    										(stop.tv_usec - start.tv_usec) / (float)1000000));

    status = check_if_sorted(sorted_array_reference, num_elements);
    if (status == -1) {
        printf("Error sorting the input array using the reference code\n");
        exit(EXIT_FAILURE);
    }

    printf("Counting sort was successful on the CPU\n");

#ifdef DEBUG
    print_array(sorted_array_reference, num_elements);
#endif

    /* FIXME: Write function to sort elements in the array in parallel fashion.
     * The result should be placed in sorted_array_mt. */
    printf("\nSorting array on GPU\n");
    sorted_array_d = (int *) malloc(num_elements * sizeof(int));
    if (sorted_array_d == NULL) {
        perror("malloc");
        exit(EXIT_FAILURE);
    }
    memset(sorted_array_d, 0, num_elements);
    compute_on_device(input_array, sorted_array_d, num_elements, range);

    /* Check the two results for correctness */
    printf("\nComparing CPU and GPU results\n");
    status = compare_results(sorted_array_reference, sorted_array_d, num_elements);
    printf(sorted_array_reference);
    printf(sorted_array_d);
    printf(num_elements);
    printf(status);
    if (status == 0)
        printf("Test passed\n");
    else
        printf("Test failed\n");

    exit(EXIT_SUCCESS);
}


/* FIXME: Write the GPU implementation of counting sort */
void compute_on_device(int *input_array, int *sorted_array, int num_elements, int range)
{
  int *device_input = NULL;
  int *device_output = NULL;

  /* initializing historgram step variables */
  int *device_hist = NULL;

  int *histogram = (int *) malloc(sizeof(int) * HISTOGRAM_SIZE);
  memset(histogram, 0, HISTOGRAM_SIZE);


  /* initializing prefix scan variables */
  int *device_scan = NULL;

  int *scan_out = (int *) malloc(sizeof(int) * HISTOGRAM_SIZE);
  memset(scan_out, 0 , sizeof(int) * HISTOGRAM_SIZE);



  /* Allocating space on GPU */
  hipMalloc((void**) &device_input, num_elements * sizeof(int));
  hipMalloc((void**) &device_output, num_elements * sizeof(int));
  hipMalloc((void**) &device_hist, HISTOGRAM_SIZE * sizeof(int));
  hipMalloc((void**) &device_scan, HISTOGRAM_SIZE * sizeof(int));


  /* CUDA copying memory from host to device */
  hipMemcpy(device_output, sorted_array, num_elements * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_input, input_array, num_elements * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_hist, histogram, HISTOGRAM_SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_scan, scan_out, HISTOGRAM_SIZE * sizeof(int), hipMemcpyHostToDevice);

  /* Kernel Configuration */
  dim3 thread_block(THREAD_BLOCK_SIZE, 1, 1);
  dim3 grid(NUM_BLOCKS, 1);

  struct timeval start, stop;

  gettimeofday(&start, NULL);
  /* Histogram generation */
  kernel_histogram<<<grid, thread_block>>>(device_input, device_hist, num_elements, HISTOGRAM_SIZE);
  hipDeviceSynchronize();
  check_for_error("Kernel Failure: Histogram");

  /* Kernel Configuration for Scanning */
  grid.x = 1;
  thread_block.x = HISTOGRAM_SIZE;

  int shared_memory_size = sizeof(int) * HISTOGRAM_SIZE;

  kernel_scan<<<grid, thread_block, 2 * shared_memory_size>>>(device_scan, device_hist, HISTOGRAM_SIZE);
  hipDeviceSynchronize();
  check_for_error("Kernel Failure: Prefix Scanning");

  /* Kernel Configuration for Count Sort */
  grid.x = NUM_BLOCKS;
  thread_block.x  = THREAD_BLOCK_SIZE;

  kernel_counting_sort<<<grid, thread_block>>>(device_output, device_scan, HISTOGRAM_SIZE);
  hipDeviceSynchronize();
  check_for_error("Kernel Failure: Counting Sort");
  gettimeofday(&stop, NULL);

  /* Copying results back from GPU */
  hipMemcpy(sorted_array, device_output, num_elements * sizeof(int), hipMemcpyDeviceToHost);

  fprintf(stderr, "Execution time for GPU = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                      (stop.tv_usec - start.tv_usec) / (float)1000000));

  /* Freeing memory on GPU/ Clean up device memory */
  hipFree(device_input);
  hipFree(device_output);
  hipFree(device_hist);
  hipFree(device_scan);

  free(histogram);
  free(scan_out);

  return;
}

/* Check for errors for CUDA run time during Kernel execution */
void check_for_error(const char *msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
    printf("CUDA ERROR: %s (%s)\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return;
}

/* Check if array is sorted */
int check_if_sorted(int *array, int num_elements)
{
    int status = 0;
    int i;
    for (i = 1; i < num_elements; i++) {
        if (array[i - 1] > array[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Check if the arrays elements are identical */
int compare_results(int *array_1, int *array_2, int num_elements)
{
    int status = 0;
    int i;
    for (i = 0; i < num_elements; i++) {
        if (array_1[i] != array_2[i]) {
            status = -1;
            break;
        }
    }

    return status;
}

/* Return random integer between [min, max] */
int rand_int(int min, int max)
{
    float r = rand()/(float)RAND_MAX;
    return (int)floorf(min + (max - min) * r);
}

/* Print given array */
void print_array(int *this_array, int num_elements)
{
    printf("Array: ");
    int i;
    for (i = 0; i < num_elements; i++)
        printf("%d ", this_array[i]);

    printf("\n");
    return;
}

/* Return min and max values in given array */
void print_min_and_max_in_array(int *this_array, int num_elements)
{
    int i;

    int current_min = INT_MAX;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] < current_min)
            current_min = this_array[i];

    int current_max = INT_MIN;
    for (i = 0; i < num_elements; i++)
        if (this_array[i] > current_max)
            current_max = this_array[i];

    printf("Minimum value in the array = %d\n", current_min);
    printf("Maximum value in the array = %d\n", current_max);
    return;
}
